#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

const int redux = 3;
const int frame_width = 640;
const int frame_height = 360;

__global__ void cudaProcess(cv::Vec3b* input, cv::Vec3b* output, int width, int redux) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int col = bx * blockDim.x * redux + tx * redux;
    int row = by * blockDim.y * redux + ty * redux;

    int red = 0, green = 0, blue = 0;

    for (int i = 0; i < redux; i++) {
        for (int j = 0; j < redux; j++) {
            blue += static_cast<int>(input[(row + i) * width + col + j].val[0]);
            green += static_cast<int>(input[(row + i) * width + col + j].val[1]);
            red += static_cast<int>(input[(row + i) * width + col + j].val[2]);
        }
    }

    int outputCol = bx * blockDim.x + tx;
    int outputRow = by * blockDim.y + ty;

    output[outputRow * (width / redux) + outputCol].val[0] = blue / (redux * redux);
    output[outputRow * (width / redux) + outputCol].val[1] = green / (redux * redux);
    output[outputRow * (width / redux) + outputCol].val[2] = red / (redux * redux);
}

int main(int argc, char *argv[]) {
    if(argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <input video path> <output video path> <threads per block>" << std::endl;
        return -1;
    }

    char *archive_read = argv[1];
    char *archive_write = argv[2];
    int threadsPerDim = atoi(argv[3]);

    cv::VideoCapture inputVideo(archive_read);
    if (!inputVideo.isOpened()) {
        std::cerr << "Error al abrir el archivo de video original." << std::endl;
        return -1;
    }

    if(threadsPerDim > 32) 
    {
      std::cerr << "Error. El valor debe ser menor o igual a 32. Tenga encuenta que el parametro al cuadrado define el numero de hilos por bloque." << std::endl;
      return -1;
    }

    double fps = inputVideo.get(cv::CAP_PROP_FPS);
    cv::VideoWriter outputVideo(archive_write, cv::VideoWriter::fourcc('X', '2', '6', '4'), fps, cv::Size(frame_width, frame_height));

    int originalWidth = inputVideo.get(cv::CAP_PROP_FRAME_WIDTH);
    int originalHeight = inputVideo.get(cv::CAP_PROP_FRAME_HEIGHT);

    cv::Vec3b *d_input, *d_output;
    hipMalloc(&d_input, originalWidth * originalHeight * sizeof(cv::Vec3b));
    hipMalloc(&d_output, frame_width * frame_height * sizeof(cv::Vec3b));

    dim3 threadsPerBlock(threadsPerDim, threadsPerDim);
    dim3 blocksPerGrid((frame_width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                      (frame_height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    cv::Mat frame;
    cv::Mat new_frame(frame_height, frame_width, CV_8UC3, cv::Scalar(255, 255, 255));

    // Declaración de eventos para medir tiempos
    hipEvent_t start, stop;
    float elapsedTime;

    // Inicializar los eventos para medir tiempos
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Registrar el evento de inicio
    hipEventRecord(start, 0);

    while (inputVideo.read(frame)) {
      hipMemcpyAsync(d_input, frame.data, originalWidth * originalHeight * sizeof(cv::Vec3b), hipMemcpyHostToDevice);

      cudaProcess<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, originalWidth, redux); // Añadido el argumento redux

      hipMemcpyAsync(new_frame.data, d_output, frame_width * frame_height * sizeof(cv::Vec3b), hipMemcpyDeviceToHost);

      outputVideo.write(new_frame);
  }


    // Registrar el evento de finalización
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Tiempo total de procesamiento: " << elapsedTime / 1000.0 << " segundos" << std::endl;

    // Limpieza
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_input);
    hipFree(d_output);

    inputVideo.release();
    outputVideo.release();

    return 0;
}

//Compila el programa
//!nvcc -w -o ./sehernandezce-lumendezm/src/video_reduction_cuda ./sehernandezce-lumendezm/src/video_reduction_cuda.cu `pkg-config --cflags --libs opencv4` -std=c++11

// Ejecuta el programa para 32*32 = 1024 hilos por bloque
//!time -p ./sehernandezce-lumendezm/src/video_reduction_cuda ./sehernandezce-lumendezm/media/inputVideo.mp4 ./sehernandezce-lumendezm/media/outputVideo.mp4 32